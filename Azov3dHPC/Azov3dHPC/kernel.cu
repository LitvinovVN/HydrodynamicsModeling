#include "hip/hip_runtime.h"
#include "header.h"
#include "source.cpp"

#pragma region ���������� ����������

int n1 = 355;// ���������� ����� �� ��� X
int n2 = 233;// ���������� ����� �� ��� Y
int n3 = 36;// ���������� ����� �� ��� Z

int n = n1 * n2 * n3;// ���������� �����
int n2d = n1 * n2;// ���������� ����� � ��������� XY

double      dt = 500;// ��� �� ������� ???????
double      t = 0;// ������ ������������� �� �������
double      dx = 1000;//6800/n1;// ��� �� X ????
double      dy = 1000;//280/n2; // ��� �� Y ????
double      dz = 0.5; //280/n2; // ��� �� Z ????
double      t0 = 1000;//1000000; // ��������� ������� �������
double      pa = 1.29;// ��������� ���������
double      po = 1000;// ��������� ������ �����
double      pv = 2700;// ��������� ������
double      ev = 0.00000000001;// ???
double      ep = 0.00000000001;// ??
double      ek = 0.000001;// ??
double      kor = 0.000103;// ����������� ���� ���������
double      cp_v = 0.0088;//????
double      cp_a = 0.0026;//????
double      g = 9.81;// ��������� ��������� �������

double      sk_u = 0;//-0.1; ���������� �������� ����� (�������� �������� ����� �� ��� X)
double      sk_v = -40;// ���������� �������� ����� (�������� �������� ����� �� ��� Y)

double      ca = pa * cp_a / po;// ����������� ���� ������ � ������
double      cv = pv * cp_v / po;// ����������� ���� ������ � ���
double      isp = 606 / (38869 * dx * dy);// ����������� ���������
#pragma endregion

#pragma region CUDA-����
__global__ void addKernel(int* c, const int* a, const int* b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}
#pragma endregion

int main()
{
    double* u = new double[n];// ������ �������� ������� �������� ������� ������ �� ��� X
    double* v = new double[n];// ������ �������� ������� �������� ������� ������ �� ��� Y
    double* w = new double[n];// ������ �������� ������� �������� ������� ������ �� ��� Z
    double* a = new double[n];// ������ �������� �������� (���� ��������)
    double* a1 = new double[n2d];// ������ �������� ����������������� �������� (���� ����������������� ��������)
    double* mu = new double[n];// ������ �������� ������������ ������������ ���� ������������ ������������� ������
    double* nu = new double[n];// ������ �������� �������������� ������������ ���� ������������ ������������� ������
    double* o_max = new double[n];//????????????????????????????????????????????????
    double* o = new double[n];// ????
    double* h = new double[n2d];// ������ �������� �������

    int* s1 = new int[n];// ������ �������� ���� ������ (0 - ������ �� ������ � ��������� �������, 1 - ������ ������ � ��������� �������, 2 - ??????)

    int k1;

    printf("mass OK\n");

    vvod(u, v, w, a, a1, mu, nu, o, o_max, h, s1);
    printf("vvod OK\n");

	do
	{
		vvodmask(u, v, w, a, o, o_max, s1);

		firstfunc(u, v, w, mu, nu, o, s1);
		secondfunc2d(u, v, w, a, a1, o, s1);
		secondfunc(u, v, w, a, a1, o, s1);
		thirdfunc(u, v, w, a, o, s1);

		t = t + dt;

		vivod(u, v, w, a);
		printf("vivod OK\n");

	} while (t < t0);


	delete[] u;
	delete[] v;
	delete[] w;
	delete[] a;
	delete[] a1;
	delete[] s1;
	delete[] mu;
	delete[] nu;
	delete[]o_max;
	delete[]o;
	delete[]h;

	printf("del OK\n");

















    /////////////////////////////////////////////
    const int arraySize = 5;
    const int aa[arraySize] = { 1, 2, 3, 4, 5 };
    const int b[arraySize] = { 10, 20, 30, 40, 50 };
    int c[arraySize] = { 0 };

    // Add vectors in parallel.
    hipError_t cudaStatus = addWithCuda(c, aa, b, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
        c[0], c[1], c[2], c[3], c[4]);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int* c, const int* a, const int* b, unsigned int size)
{
    int* dev_a = 0;
    int* dev_b = 0;
    int* dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel << <1, size >> > (dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);

    return cudaStatus;
}



/// <summary>
/// ���� ��������� �������
/// </summary>
/// <param name="u"></param>
/// <param name="v"></param>
/// <param name="w"></param>
/// <param name="a"></param>
/// <param name="a1"></param>
/// <param name="mu"></param>
/// <param name="nu"></param>
/// <param name="o"></param>
/// <param name="o_max"></param>
/// <param name="h"></param>
/// <param name="s"></param>

void vvod(double* u,
	double* v,
	double* w,
	double* a,
	double* a1,
	double* mu,
	double* nu,
	double* o,
	double* o_max,
	double* h,
	int* s)
{
	int n = n1 * n2 * n3;
	int i, j, k, m;
	float buf;
	int m0, m2, m4, m6,
		m24, m26, m46, m246;

	double k0, k1, k2;
	//------------------------------------------------------------------------------------------------
	//								���� ������(������)
	//------------------------------------------------------------------------------------------------

	//------------------------------------------------------------------------------------------------
	//								��������� �������
	//------------------------------------------------------------------------------------------------

	for (m0 = 0; m0 < n; m0++)
	{
		u[m0] = 0;
		v[m0] = 0;
		w[m0] = 0;
		a[m0] = 0;
		mu[m0] = 1000;
		nu[m0] = 0.02;
	}

	for (j = 0; j < n2; j++)
		for (i = 0; i < n1; i++)
		{
			m = j + n2 * i;
			a1[m] = 0;
		}

	printf("nul OK\n");


	FILE* outs1;
	FILE* outs2;
	FILE* outs3;
	FILE* outs4;
	FILE* outs5;

	outs1 = fopen("setka1000.txt", "r+");
	outs2 = fopen("u.txt", "r+");
	outs3 = fopen("v.txt", "r+");
	outs4 = fopen("w.txt", "r+");
	outs5 = fopen("a.txt", "r+");



	for (k = 0; k < n3; k++)
		for (j = 0; j < n2; j++)
			for (i = 0; i < n1; i++)
			{
				m0 = k + n3 * j + n2 * n3 * i;

				o[m0] = 0;
				o_max[m0] = 0;
			}



	for (j = 0; j < n2 - 1; j++)
		for (i = 0; i < n1 - 1; i++)
		{
			m = j + n2 * i;

			fscanf(outs1, "%f", &buf);
			h[m] = buf;
		}


	for (k = 0; k < n3; k++)
		for (j = 0; j < n2; j++)
			for (i = 0; i < n1; i++)
			{
				m0 = k + n3 * j + n2 * n3 * i;
				m = j + n2 * i;


				if (h[m] > 0)
				{
					o_max[m0] = 2 * h[m] + 6 - k;
					if (o_max[m0] > 1)
						o_max[m0] = 1;
					if (o_max[m0] < 0)
						o_max[m0] = 0;
					if (k < 6)
						o_max[m0] = 1;

					if (k > 5)
						o[m0] = o_max[m0];
				}

			}


	for (k = 1; k < n3 - 1; k++)
		for (i = 1; i < n1 - 1; i++)
			for (j = 1; j < n2 - 1; j++)
			{
				m0 = k + n3 * j + n2 * n3 * i;
				m2 = m0 - n2 * n3;
				m4 = m0 - n3;
				m6 = m0 - 1;
				m24 = m0 - n2 * n3 - n3;
				m26 = m0 - n2 * n3 - 1;
				m46 = m0 - n3 - 1;
				m246 = m0 - n2 * n3 - n3 - 1;

				k1 = 0.25 * (o[m0] + o[m4] + o[m6] + o[m46]);
				k2 = 0.25 * (o[m2] + o[m24] + o[m26] + o[m246]);
				k0 = 0.5 * (k1 + k2);

				s[m0] = 0;
				if (k0 > 0)
					s[m0] = 1;


			}
	printf("h OK\n");




	for (k = 0; k < n3; k++)
		for (j = 0; j < n2; j++)
			for (i = 0; i < n1; i++)
			{
				m0 = k + n3 * j + n2 * n3 * i;

				//fscanf(outs2, "%f", &buf);
				//u[m0]=buf;
				//fscanf(outs3, "%f", &buf);
				//v[m0]=buf;
				//fscanf(outs4, "%f", &buf);
				//w[m0]=buf;
				//fscanf(outs5, "%f", &buf);
				//a[m0]=buf;

			}
	fclose(outs1);
	//fclose(outs2);
	//fclose(outs3);
	//fclose(outs4);
	//fclose(outs5);


	//------------------------------------------------------------------------------------------------
	//								��������� �������(�����)
	//------------------------------------------------------------------------------------------------

	//------------------------------------------------------------------------------------------------
	//								���� ������(�����)
	//------------------------------------------------------------------------------------------------
}





void vvodmask(double* u, double* v, double* w,
	double* a, double* o, double* o_max, int* s)

{
	int i, j, k;
	int m0, m2, m4, m5, m6,
		m24, m26, m46, m246, m15, m35, m135;

	double k0, k1, k2, k3, k4;
	double sum = 0;


	for (k = 1; k < n3 - 2; k++)
		for (j = 1; j < n2 - 2; j++)
			for (i = 1; i < n1 - 2; i++)
			{
				m0 = k + n3 * j + n2 * n3 * i;
				m5 = m0 + 1;
				m15 = m0 + n2 * n3 + 1;
				m35 = m0 + n3 + 1;
				m135 = m0 + n2 * n3 + n3 + 1;


				o[m0] = (a[m5] + a[m15] + a[m35] + a[m135]) / (4 * po * g * dz) + k - 5;// ???????????????????????????????????????????????????????

				if (o[m0] > o_max[m0])
					o[m0] = o_max[m0];

				if (o[m0] < 0)
					o[m0] = 0;
			}





	for (k = 1; k < n3 - 1; k++)
		for (i = 1; i < n1 - 1; i++)
			for (j = 1; j < n2 - 1; j++)
			{
				m0 = k + n3 * j + n2 * n3 * i;
				m2 = m0 - n2 * n3;
				m4 = m0 - n3;
				m6 = m0 - 1;
				m24 = m0 - n2 * n3 - n3;
				m26 = m0 - n2 * n3 - 1;
				m46 = m0 - n3 - 1;
				m246 = m0 - n2 * n3 - n3 - 1;

				k1 = 0.25 * (o[m0] + o[m4] + o[m6] + o[m46]);
				k2 = 0.25 * (o[m2] + o[m24] + o[m26] + o[m246]);
				k0 = 0.5 * (k1 + k2);

				if ((k0 > 0) && (s[m0] == 0))
					a[m0] = po * g * dz * (5 - k); // ???????????????????�� �������� � �������

				s[m0] = 0;
				if (k0 > 0)
					s[m0] = 1;
				if (k0 == 0)
				{
					a[m0] = 0;
					u[m0] = 0;
					v[m0] = 0;
					w[m0] = 0;
				}

			}

	j = 132;    // c����
	i = 1;
	for (k = 1; k < n3; k++)
	{
		m0 = k + n3 * j + n2 * n3 * i;
		m2 = m0 - n2 * n3;
		m4 = m0 - n3;
		m6 = m0 - 1;
		m24 = m0 - n2 * n3 - n3;
		m26 = m0 - n2 * n3 - 1;
		m46 = m0 - n3 - 1;
		m246 = m0 - n2 * n3 - n3 - 1;

		k1 = 0.25 * (o[m0] + o[m4] + o[m6] + o[m46]);
		k2 = 0.25 * (o[m2] + o[m24] + o[m26] + o[m246]);
				
		sum = sum + k1 - k2;//???????????????????????????????????????????
	}


	for (k = 1; k < n3; k++)
	{
		m0 = k + n3 * j + n2 * n3 * i;

		if (s[m0] > 0)
		{
			u[m0] = -115 / (dy * dz * sum);//???????????????????????????
			v[m0] = 0;
			s[m0] = 2;
		}
	}


	sum = 0;
	j = 6;    // ����� ������� ����� � ������� ��������
	i = 348;
	for (k = 1; k < n3; k++)
	{
		m0 = k + n3 * j + n2 * n3 * i;
		m2 = m0 - n2 * n3;
		m4 = m0 - n3;
		m6 = m0 - 1;
		m24 = m0 - n2 * n3 - n3;
		m26 = m0 - n2 * n3 - 1;
		m46 = m0 - n3 - 1;
		m246 = m0 - n2 * n3 - n3 - 1;

		k1 = 0.25 * (o[m0] + o[m4] + o[m6] + o[m46]);
		k2 = 0.25 * (o[m2] + o[m24] + o[m26] + o[m246]);

		sum = sum + k2 - k1;
	}


	for (k = 1; k < n3; k++)
	{
		m0 = k + n3 * j + n2 * n3 * i;

		if (s[m0] > 0)
		{
			u[m0] = -390 / (dy * dz * sum);
			v[m0] = 0;
			s[m0] = 2;
		}
	}


	sum = 0;
	j = 10;    // ����� ��������
	i = 347;
	for (k = 1; k < n3; k++)
	{
		m0 = k + n3 * j + n2 * n3 * i;
		m2 = m0 - n2 * n3;
		m4 = m0 - n3;
		m6 = m0 - 1;
		m24 = m0 - n2 * n3 - n3;
		m26 = m0 - n2 * n3 - 1;
		m46 = m0 - n3 - 1;
		m246 = m0 - n2 * n3 - n3 - 1;

		k1 = 0.25 * (o[m0] + o[m4] + o[m6] + o[m46]);
		k2 = 0.25 * (o[m2] + o[m24] + o[m26] + o[m246]);

		sum = sum + k2 - k1;
	}


	for (k = 1; k < n3; k++)
	{
		m0 = k + n3 * j + n2 * n3 * i;

		if (s[m0] > 0)
		{
			u[m0] = -424 / (dy * dz * sum);
			v[m0] = 0;
			s[m0] = 2;
		}
	}


	sum = 0;
	j = 16;    // ����� ������ ��������
	i = 348;
	for (k = 1; k < n3; k++)
	{
		m0 = k + n3 * j + n2 * n3 * i;
		m2 = m0 - n2 * n3;
		m4 = m0 - n3;
		m6 = m0 - 1;
		m24 = m0 - n2 * n3 - n3;
		m26 = m0 - n2 * n3 - 1;
		m46 = m0 - n3 - 1;
		m246 = m0 - n2 * n3 - n3 - 1;

		k1 = 0.25 * (o[m0] + o[m4] + o[m6] + o[m46]);
		k2 = 0.25 * (o[m2] + o[m24] + o[m26] + o[m246]);

		sum = sum + k2 - k1;
	}


	for (k = 1; k < n3; k++)
	{
		m0 = k + n3 * j + n2 * n3 * i;

		if (s[m0] > 0)
		{
			u[m0] = -185 / (dy * dz * sum);
			v[m0] = 0;
			s[m0] = 2;
		}
	}


	sum = 0;
	j = 19;    // ����� ���������
	i = 348;
	for (k = 1; k < n3; k++)
	{
		m0 = k + n3 * j + n2 * n3 * i;
		m2 = m0 - n2 * n3;
		m4 = m0 - n3;
		m6 = m0 - 1;
		m24 = m0 - n2 * n3 - n3;
		m26 = m0 - n2 * n3 - 1;
		m46 = m0 - n3 - 1;
		m246 = m0 - n2 * n3 - n3 - 1;

		k1 = 0.25 * (o[m0] + o[m4] + o[m6] + o[m46]);
		k2 = 0.25 * (o[m2] + o[m24] + o[m26] + o[m246]);

		sum = sum + k2 - k1;
	}


	for (k = 1; k < n3; k++)
	{
		m0 = k + n3 * j + n2 * n3 * i;

		if (s[m0] > 0)
		{
			u[m0] = -105 / (dy * dz * sum);
			v[m0] = 0;
			s[m0] = 2;
		}
	}


	sum = 0;
	j = 22;    // ����� ��������
	i = 350;
	for (k = 1; k < n3; k++)
	{
		m0 = k + n3 * j + n2 * n3 * i;
		m2 = m0 - n2 * n3;
		m4 = m0 - n3;
		m6 = m0 - 1;
		m24 = m0 - n2 * n3 - n3;
		m26 = m0 - n2 * n3 - 1;
		m46 = m0 - n3 - 1;
		m246 = m0 - n2 * n3 - n3 - 1;

		k1 = 0.25 * (o[m0] + o[m4] + o[m6] + o[m46]);
		k2 = 0.25 * (o[m2] + o[m24] + o[m26] + o[m246]);

		sum = sum + k2 - k1;
	}


	for (k = 1; k < n3; k++)
	{
		m0 = k + n3 * j + n2 * n3 * i;

		if (s[m0] > 0)
		{
			u[m0] = -199 / (dy * dz * sum);
			v[m0] = 0;
			s[m0] = 2;
		}
	}


	sum = 0;
	j = 29;    // ����� ������, ������ � ������
	i = 353;
	for (k = 1; k < n3; k++)
	{
		m0 = k + n3 * j + n2 * n3 * i;
		m2 = m0 - n2 * n3;
		m4 = m0 - n3;
		m6 = m0 - 1;
		m24 = m0 - n2 * n3 - n3;
		m26 = m0 - n2 * n3 - 1;
		m46 = m0 - n3 - 1;
		m246 = m0 - n2 * n3 - n3 - 1;

		k1 = 0.25 * (o[m0] + o[m4] + o[m6] + o[m46]);
		k2 = 0.25 * (o[m2] + o[m24] + o[m26] + o[m246]);

		sum = sum + k2 - k1;
	}


	for (k = 1; k < n3; k++)
	{
		m0 = k + n3 * j + n2 * n3 * i;

		if (s[m0] > 0)
		{
			u[m0] = -82 / (dy * dz * sum);
			v[m0] = 0;
			s[m0] = 2;
		}
	}


	sum = 0;
	j = 223;    // ������
	i = 207;
	for (k = 1; k < n3; k++)
	{
		m0 = k + n3 * j + n2 * n3 * i;
		m2 = m0 - n2 * n3;
		m4 = m0 - n3;
		m6 = m0 - 1;
		m24 = m0 - n2 * n3 - n3;
		m26 = m0 - n2 * n3 - 1;
		m46 = m0 - n3 - 1;
		m246 = m0 - n2 * n3 - n3 - 1;

		k3 = 0.25 * (o[m0] + o[m2] + o[m6] + o[m26]);
		k4 = 0.25 * (o[m4] + o[m24] + o[m46] + o[m246]);

		sum = sum + k4 - k3;
	}


	for (k = 1; k < n3; k++)
	{
		m0 = k + n3 * j + n2 * n3 * i;

		if (s[m0] > 0)
		{
			u[m0] = 0;
			v[m0] = -923 / (dx * dz * sum);
			s[m0] = 2;
		}
	}

	j = 231;
	for (i = 1; i < n1; i++)
		for (k = 1; k < n3; k++)
		{
			m0 = k + n3 * j + n2 * n3 * i;
			m2 = m0 - n2 * n3;
			m4 = m0 - n3;
			m6 = m0 - 1;
			m24 = m0 - n2 * n3 - n3;
			m26 = m0 - n2 * n3 - 1;
			m46 = m0 - n3 - 1;
			m246 = m0 - n2 * n3 - n3 - 1;

			k3 = 0.25 * (o[m0] + o[m2] + o[m6] + o[m26]);
			k4 = 0.25 * (o[m4] + o[m24] + o[m46] + o[m246]);
			k0 = 0.5 * (k3 + k4);

			if (k4 > k3)
			{
				s[m0] = 3;
			}
		}
}




void vivod(double* u, double* v, double* w, double* a)
{
	int i, j, k, m0;

	FILE* uoutfile;
	FILE* voutfile;
	FILE* woutfile;
	FILE* aoutfile;



	uoutfile = fopen("u.txt", "w+");
	voutfile = fopen("v.txt", "w+");
	woutfile = fopen("w.txt", "w+");
	aoutfile = fopen("a.txt", "w+");


	for (k = 0; k < n3; k++)
	{
		for (j = 0; j < n2; j++)
		{
			for (i = 0; i < n1; i++)
			{
				m0 = k + n3 * j + n2 * n3 * i;
				fprintf(uoutfile, "%f	", u[m0]);
				fprintf(voutfile, "%f	", v[m0]);
				fprintf(woutfile, "%f	", w[m0]);
				fprintf(aoutfile, "%f	", a[m0]);

			}
			fprintf(uoutfile, "\n");
			fprintf(voutfile, "\n");
			fprintf(woutfile, "\n");
			fprintf(aoutfile, "\n");

		}
	}
	fclose(uoutfile);
	fclose(voutfile);
	fclose(woutfile);
	fclose(aoutfile);

}

//------------------------------------------------------------------------------------------------
//										1
//									   11		 ������ 
//									  1 1		�������� �������
//										1		�� �������������
//										1		��������� ����.
//										1		(��������� ��������-
//									  11111		���������)
//------------------------------------------------------------------------------------------------

void firstfunc(double* u, double* v, double* w,
	double* mu, double* nu, double* o, int* s)
{
	double omega, sk;
	int i, j, k, it = 0;
	int m0, m1, m2, m3, m4, m5, m6,
		m24, m26, m46, m246;

	double k0, k1, k2, k3, k4, k5, k6;

	double* c0 = new double[n];
	double* c1 = new double[n];
	double* c2 = new double[n];
	double* c3 = new double[n];
	double* c4 = new double[n];
	double* c5 = new double[n];
	double* c6 = new double[n];

	double* f1 = new double[n];
	double* f2 = new double[n];
	double* f3 = new double[n];



	for (m0 = 0; m0 < n; m0++)
	{
		c1[m0] = 0;
		c2[m0] = 0;
		c3[m0] = 0;
		c4[m0] = 0;
		c5[m0] = 0;
		c6[m0] = 0;
	}
	printf("mass1 OK\n");
	//------------------------------------------------------------------------------------------------
	//								����������� � ������ �����
	//------------------------------------------------------------------------------------------------

	for (k = 1; k < n3 - 1; k++)
		for (i = 1; i < n1 - 1; i++)
			for (j = 1; j < n2 - 1; j++)
			{
				m0 = k + n3 * j + n2 * n3 * i;
				m1 = m0 + n2 * n3;
				m2 = m0 - n2 * n3;
				m3 = m0 + n3;
				m4 = m0 - n3;
				m5 = m0 + 1;
				m6 = m0 - 1;
				m24 = m0 - n2 * n3 - n3;
				m26 = m0 - n2 * n3 - 1;
				m46 = m0 - n3 - 1;
				m246 = m0 - n2 * n3 - n3 - 1;

				k1 = 0.25 * (o[m0] + o[m4] + o[m6] + o[m46]);
				k2 = 0.25 * (o[m2] + o[m24] + o[m26] + o[m246]);
				k3 = 0.25 * (o[m0] + o[m2] + o[m6] + o[m26]);
				k4 = 0.25 * (o[m4] + o[m24] + o[m46] + o[m246]);
				k5 = 0.25 * (o[m0] + o[m2] + o[m4] + o[m24]);
				k6 = 0.25 * (o[m6] + o[m26] + o[m46] + o[m246]);
				k0 = 0.5 * (k1 + k2);


				c1[m0] = 0.5 * dt * (-(u[m1] + u[m0]) / 2 + (mu[m1] + mu[m0]) / dx) / dx;
				c2[m0] = 0.5 * dt * ( (u[m2] + u[m0]) / 2 + (mu[m2] + mu[m0]) / dx) / dx;
				c3[m0] = 0.5 * dt * (-(v[m3] + v[m0]) / 2 + (mu[m3] + mu[m0]) / dy) / dy;
				c4[m0] = 0.5 * dt * ( (v[m4] + v[m0]) / 2 + (mu[m4] + mu[m0]) / dy) / dy;
				c5[m0] = 0.5 * dt * (-(w[m5] + w[m0]) / 2 + (nu[m5] + nu[m0]) / dz) / dz;
				c6[m0] = 0.5 * dt * ( (w[m6] + w[m0]) / 2 + (nu[m6] + nu[m0]) / dz) / dz;


				if ((c1[m0] < 0) && (k1 > 0))
				{
					c1[m0] = 0.5 * dt * ((mu[m1] + mu[m0]) / dx) / dx;
					c2[m0] = 0.5 * dt * ((u[m2] + u[m0]) + (mu[m2] + mu[m0]) / dx) / dx;
				}
				if ((c2[m0] < 0) && (k2 > 0))
				{
					c1[m0] = 0.5 * dt * (-(u[m1] + u[m0]) + (mu[m1] + mu[m0]) / dx) / dx;
					c2[m0] = 0.5 * dt * ((mu[m2] + mu[m0]) / dx) / dx;
				}
				if ((c3[m0] < 0) && (k3 > 0))
				{
					c3[m0] = 0.5 * dt * ((mu[m3] + mu[m0]) / dy) / dy;
					c4[m0] = 0.5 * dt * ((v[m4] + v[m0]) + (mu[m4] + mu[m0]) / dy) / dy;
				}
				if ((c4[m0] < 0) && (k4 > 0))
				{
					c3[m0] = 0.5 * dt * (-(v[m3] + v[m0]) + (mu[m3] + mu[m0]) / dy) / dy;
					c4[m0] = 0.5 * dt * ((mu[m4] + mu[m0]) / dy) / dy;
				}
				if ((c5[m0] < 0) && (k5 > 0))
				{
					c5[m0] = 0.5 * dt * ((nu[m5] + nu[m0]) / dz) / dz;
					c6[m0] = 0.5 * dt * ((w[m6] + w[m0]) + (nu[m6] + nu[m0]) / dz) / dz;
				}
				if ((c6[m0] < 0) && (k6 > 0))
				{
					c5[m0] = 0.5 * dt * (-(w[m5] + w[m0]) + (nu[m5] + nu[m0]) / dz) / dz;
					c6[m0] = 0.5 * dt * ((nu[m6] + nu[m0]) / dz) / dz;
				}

				c1[m0] = k1 * c1[m0];
				c2[m0] = k2 * c2[m0];
				c3[m0] = k3 * c3[m0];
				c4[m0] = k4 * c4[m0];
				c5[m0] = k5 * c5[m0];
				c6[m0] = k6 * c6[m0];


				c0[m0] = k0 + c1[m0] + c3[m0] + c5[m0] + c2[m0] + c4[m0] + c6[m0];

				f1[m0] = k0 * (u[m0] + dt * kor * (v[m0] - w[m0]));
				f2[m0] = k0 * (v[m0] - dt * kor * u[m0]);
				f3[m0] = k0 * (w[m0] + dt * kor * u[m0]);


				if (k5 > k6)
				{
					sk = dt * ca * sqrt((sk_u - u[m0]) * (sk_u - u[m0]) + (sk_v - v[m0]) * (sk_v - v[m0]) + w[m0] * w[m0]);
					f1[m0] = f1[m0] + (k5 - k6) * sk * (sk_u) / dz;
					f2[m0] = f2[m0] + (k5 - k6) * sk * (sk_v) / dz;
					c0[m0] = c0[m0] + (k5 - k6) * sk / dz;

				}

				if (k6 > k5)
				{
					sk = dt * cv * sqrt(u[m0] * u[m0] + v[m0] * v[m0] + w[m0] * w[m0]) / dz;
					c0[m0] = c0[m0] + (k6 - k5) * sk;
				}


				if (s[m0] == 2)
				{
					c1[m0] = 0;
					c2[m0] = 0;
					c3[m0] = 0;
					c4[m0] = 0;
					c5[m0] = 0;
					c6[m0] = 0;


					c0[m0] = 1;

					f1[m0] = u[m0];
					f2[m0] = v[m0];
					f3[m0] = w[m0];

				}


			}
	printf("func1 OK\n");
	omega = 0;

	ptmss(u, c1, c2, c3, c4, c5, c6, c0, f1, s, omega, ev);
	ptmss(v, c1, c2, c3, c4, c5, c6, c0, f2, s, omega, ev);
	//ptmss(w,c1,c2,c3,c4,c5,c6,c0,f3,s,omega,ev);


	delete[] c0;
	delete[] c1;
	delete[] c2;
	delete[] c3;
	delete[] c4;
	delete[] c5;
	delete[] c6;

	delete[] f1;
	delete[] f2;
	delete[] f3;

}

//------------------------------------------------------------------------------------------------
//									   222
//									  2   2		 �������
//									      2		�������� 
//										 2		(���������
//										2		��������)
//									   2		(����� ����������� �������)
//									  22222
//------------------------------------------------------------------------------------------------

void secondfunc(double* u, double* v, double* w, double* a, double* a1, double* o, int* s)
{

	int m, m0, m1, m2, m3, m4, m5, m6,
		m24, m26, m46, m246;

	double k0, k1, k2, k3, k4, k5, k6;
	double d0, d1, d2, d3;
	double omega;
	int i, j, k, it = 0;
	double* f = new double[n];

	double* c0 = new double[n];
	double* c1 = new double[n];
	double* c2 = new double[n];
	double* c3 = new double[n];
	double* c4 = new double[n];
	double* c5 = new double[n];
	double* c6 = new double[n];


	d0 = 2 / (dx * dx) + 2 / (dy * dy) + 2 / (dz * dz);
	d1 = 1 / (dx * dx);
	d2 = 1 / (dy * dy);
	d3 = 1 / (dz * dz);

	//------------------------------------------------------------------------------------------------
	//								������ �����
	//------------------------------------------------------------------------------------------------

	for (k = 1; k < n3 - 1; k++)
		for (i = 1; i < n1 - 1; i++)
			for (j = 1; j < n2 - 1; j++)
			{
				m0 = k + n3 * j + n2 * n3 * i;
				m1 = m0 + n2 * n3;
				m2 = m0 - n2 * n3;
				m3 = m0 + n3;
				m4 = m0 - n3;
				m5 = m0 + 1;
				m6 = m0 - 1;
				m24 = m0 - n2 * n3 - n3;
				m26 = m0 - n2 * n3 - 1;
				m46 = m0 - n3 - 1;
				m246 = m0 - n2 * n3 - n3 - 1;

				k1 = 0.25 * (o[m0] + o[m4] + o[m6] + o[m46]);
				k2 = 0.25 * (o[m2] + o[m24] + o[m26] + o[m246]);
				k3 = 0.25 * (o[m0] + o[m2] + o[m6] + o[m26]);
				k4 = 0.25 * (o[m4] + o[m24] + o[m46] + o[m246]);
				k5 = 0.25 * (o[m0] + o[m2] + o[m4] + o[m24]);
				k6 = 0.25 * (o[m6] + o[m26] + o[m46] + o[m246]);
				k0 = 0.5 * (k1 + k2);



				c1[m0] = d1 * k1;
				c2[m0] = d1 * k2;
				c3[m0] = d2 * k3;
				c4[m0] = d2 * k4;
				c5[m0] = d3 * k5;
				c6[m0] = d3 * k6;


				c0[m0] = d1 * k1 + d1 * k2 + d2 * k3 + d2 * k4 + d3 * k5 + d3 * k6;


				f[m0] = -(po / dt) * (k1 * (u[m1] + u[m0]) / (2 * dx) - k2 * (u[m2] + u[m0]) / (2 * dx)
					+ k3 * (v[m3] + v[m0]) / (2 * dy) - k4 * (v[m4] + v[m0]) / (2 * dy)
					+ k5 * (w[m5] + w[m0]) / (2 * dz) - k6 * (w[m6] + w[m0]) / (2 * dz));


				if (k5 > k6)
				{
					c0[m0] = c0[m0] + (k5 - k6) / (dt * dt * dz * g);
					f[m0] = f[m0] + (k5 - k6) * a[m0] / (dt * dt * dz * g) - (po / dt) * isp * (k5 - k6);
				}

				if (s[m0] > 1)
					f[m0] = f[m0] - (po / dt) * ((k2 - k1) * u[m0] / dx + (k4 - k3) * v[m0] / dy);

			}

	for (k = 1; k < n3 - 1; k++)
		for (i = 1; i < n1 - 1; i++)
			for (j = 1; j < n2 - 1; j++)
			{
				m = j + n2 * i;
				m0 = k + n3 * j + n2 * n3 * i;


				if (s[m0] > 0)
					a[m0] = a1[m];
			}

	printf("a2d OK\n");

	omega = 0;//1/(2*d0);

	//ptmss(a,c1,c2,c3,c4,c5,c6,c0,f,s,omega,ep);

	delete[] c0;
	delete[] c1;
	delete[] c2;
	delete[] c3;
	delete[] c4;
	delete[] c5;
	delete[] c6;


	delete[] f;

}


//------------------------------------------------------------------------------------------------
//									   333
//									  3   3		 ������
//									      3		�������� 
//									     3		�� ���������
//										  3		�������� ����.
//								      3	  3		(����� �����)
//									   333
//------------------------------------------------------------------------------------------------


void thirdfunc(double* u, double* v, double* w, double* a,
	double* o, int* s)
{
	int i, j, k;
	int m0, m1, m2, m3, m4, m5, m6,
		m24, m26, m46, m246;

	double k0, k1, k2, k3, k4, k5, k6;



	for (k = 1; k < n3 - 1; k++)
		for (i = 1; i < n1 - 1; i++)
			for (j = 1; j < n2 - 1; j++)
			{
				m0 = k + n3 * j + n2 * n3 * i;
				m1 = m0 + n2 * n3;
				m2 = m0 - n2 * n3;
				m3 = m0 + n3;
				m4 = m0 - n3;
				m5 = m0 + 1;
				m6 = m0 - 1;
				m24 = m0 - n2 * n3 - n3;
				m26 = m0 - n2 * n3 - 1;
				m46 = m0 - n3 - 1;
				m246 = m0 - n2 * n3 - n3 - 1;

				k1 = 0.25 * (o[m0] + o[m4] + o[m6] + o[m46]);
				k2 = 0.25 * (o[m2] + o[m24] + o[m26] + o[m246]);
				k3 = 0.25 * (o[m0] + o[m2] + o[m6] + o[m26]);
				k4 = 0.25 * (o[m4] + o[m24] + o[m46] + o[m246]);
				k5 = 0.25 * (o[m0] + o[m2] + o[m4] + o[m24]);
				k6 = 0.25 * (o[m6] + o[m26] + o[m46] + o[m246]);
				k0 = 0.5 * (k1 + k2);

				if ((s[m0] > 0) && (s[m0] != 2))
				{
					u[m0] = u[m0] - dt * (k1 * (a[m1] - a[m0]) - k2 * (a[m2] - a[m0])) / (2 * dx * po * k0);
					v[m0] = v[m0] - dt * (k3 * (a[m3] - a[m0]) - k4 * (a[m4] - a[m0])) / (2 * dy * po * k0);
					//  w[m0]=w[m0]-dt*(k5*(a[m5]-a[m0])-k6*(a[m6]-a[m0]))/(2*dz*po*k0);
				}

			}
}




void ptmss(double* u, double* c1, double* c2, double* c3, double* c4,
	double* c5, double* c6, double* c0,
	double* f, int* s, double omega, double e)
{
	double max;
	double tay;
	int i, j, k, it = 0;
	int m0, m1, m2, m3, m4, m5, m6;


	tay = 2 * omega;
	double RwRw, ww, Aww, Awr, Rr;


	double* r = new double[n];

	printf("ptmss start OK\n");
	//------------------------------------------------------------------------------------------------
	//								����� ����������� �������������(������)
	//------------------------------------------------------------------------------------------------
	for (m0 = 0; m0 < n; m0++)
	{
		r[m0] = 0;

	}

	do
	{
		//------------------------------------------------------------------------------------------------
		//								�������
		//------------------------------------------------------------------------------------------------

		Aww = 0;
		RwRw = 0;
		ww = 0;
		max = 0;

		for (k = 1; k < n3 - 1; k++)
			for (i = 1; i < n1 - 1; i++)
				for (j = 1; j < n2 - 1; j++)
				{
					m0 = k + n3 * j + n2 * n3 * i;
					if (c0[m0] > 0)
					{
						m1 = m0 + n2 * n3;
						m2 = m0 - n2 * n3;
						m3 = m0 + n3;
						m4 = m0 - n3;
						m5 = m0 + 1;
						m6 = m0 - 1;


						r[m0] = f[m0] - c0[m0] * u[m0] + (c1[m0] * u[m1] + c2[m0] * u[m2]
							+ c3[m0] * u[m3] + c4[m0] * u[m4] + c5[m0] * u[m5] + c6[m0] * u[m6]);
						if (max < fabs(r[m0])) { max = fabs(r[m0]); };
					}
				}

		//------------------------------------------------------------------------------------------------
		//								�������� ���������� ����������� �������
		//------------------------------------------------------------------------------------------------


		for (k = 1; k < n3 - 1; k++)
			for (i = 1; i < n1 - 1; i++)
				for (j = 1; j < n2 - 1; j++)
				{
					m0 = k + n3 * j + n2 * n3 * i;
					if (c0[m0] > 0)
					{
						m2 = m0 - n2 * n3;
						m4 = m0 - n3;
						m6 = m0 - 1;


						r[m0] = (omega * (c2[m0] * r[m2] + c4[m0] * r[m4] + c6[m0] * r[m6]) + r[m0]) / ((0.5 * omega + 1) * c0[m0]);
					}
				}


		for (k = n3 - 2; k > 0; k--)
			for (i = n1 - 2; i > 0; i--)
				for (j = n2 - 2; j > 0; j--)
				{
					m0 = k + n3 * j + n2 * n3 * i;
					if (c0[m0] > 0)
					{
						m1 = m0 + n2 * n3;
						m3 = m0 + n3;
						m5 = m0 + 1;


						r[m0] = (omega * (c1[m0] * r[m1] + c3[m0] * r[m3] + c5[m0] * r[m5]) + r[m0] * c0[m0]) / ((0.5 * omega + 1) * c0[m0]);
					}
				}


		for (k = 1; k < n3 - 1; k++)
			for (i = 1; i < n1 - 1; i++)
				for (j = 1; j < n2 - 1; j++)
				{
					m0 = k + n3 * j + n2 * n3 * i;
					if (c0[m0] > 0)
					{
						m1 = m0 + n2 * n3;
						m2 = m0 - n2 * n3;
						m3 = m0 + n3;
						m4 = m0 - n3;
						m5 = m0 + 1;
						m6 = m0 - 1;



						Awr = 1.1 * c0[m0] * r[m0] - (c1[m0] * r[m1] + c3[m0] * r[m3] + c5[m0] * r[m5] + c2[m0] * r[m2] + c4[m0] * r[m4] + c6[m0] * r[m6]);
						Rr = 0.5 * c0[m0] * r[m0] - (c1[m0] * r[m1] + c3[m0] * r[m3] + c5[m0] * r[m5]);

						RwRw = RwRw + Rr * Rr / c0[m0];
						Aww = Aww + Awr * r[m0];
						ww = ww + c0[m0] * r[m0] * r[m0];

					}
				}

		if (ww > 0)
		{
			tay = 2 * omega + ww / Aww;
			omega = sqrt(ww / RwRw);

		}

		//------------------------------------------------------------------------------------------------
		//								��� ���
		//------------------------------------------------------------------------------------------------


		for (m0 = 0; m0 < n; m0++)
			u[m0] = u[m0] + 1 * tay * r[m0];

		it++;

		printf("%f	%f	%f	%f	%d\n", t, max * 1000000, omega, tay, it);
	} while ((max > e) && (it < 300));

	printf("ptmss OK\n");

	delete[] r;


	//------------------------------------------------------------------------------------------------
	//								����� ����������� �������������(�����)
	//------------------------------------------------------------------------------------------------

}




//------------------------------------------------------------------------------------------------
//									   222
//									  2   2		 ???????
//									      2		???????? 
//										 2		(?????????
//										2		????????)
//									   2		(????? ??????????? ???????)
//									  22222
//------------------------------------------------------------------------------------------------


void secondfunc2d(double* u, double* v, double* w, double* a, double* a1, double* o, int* s)
{

	int m, m0, m1, m2, m3, m4, m5, m6,
		m24, m26, m46, m246;

	double k0, k1, k2, k3, k4, k5, k6;

	double d0, d1, d2;
	double omega;
	int i, j, k, it = 0;
	double* f = new double[n2d];
	double* a2 = new double[n2d];

	double* c0 = new double[n2d];
	double* c1 = new double[n2d];
	double* c2 = new double[n2d];
	double* c3 = new double[n2d];
	double* c4 = new double[n2d];
	double* c5 = new double[n2d];
	double* c6 = new double[n2d];

	printf("mass2d OK\n");

	d0 = 2 / (dx * dx) + 2 / (dy * dy) + 2 / (dz * dz);
	d1 = 1 / (dx * dx);
	d2 = 1 / (dy * dy);

	//------------------------------------------------------------------------------------------------
	//								?????? ?????
	//------------------------------------------------------------------------------------------------
	for (m0 = 0; m0 < n2d; m0++)
	{
		c1[m0] = 0;
		c2[m0] = 0;
		c3[m0] = 0;
		c4[m0] = 0;


		c0[m0] = 0;

		f[m0] = 0;
		a2[m0] = a1[m0];

	}


	for (k = 1; k < n3 - 1; k++)
		for (i = 1; i < n1 - 1; i++)
			for (j = 1; j < n2 - 1; j++)
			{
				m = j + n2 * i;


				m0 = k + n3 * j + n2 * n3 * i;
				m1 = m0 + n2 * n3;
				m2 = m0 - n2 * n3;
				m3 = m0 + n3;
				m4 = m0 - n3;
				m5 = m0 + 1;
				m6 = m0 - 1;
				m24 = m0 - n2 * n3 - n3;
				m26 = m0 - n2 * n3 - 1;
				m46 = m0 - n3 - 1;
				m246 = m0 - n2 * n3 - n3 - 1;

				k1 = 0.25 * (o[m0] + o[m4] + o[m6] + o[m46]);
				k2 = 0.25 * (o[m2] + o[m24] + o[m26] + o[m246]);
				k3 = 0.25 * (o[m0] + o[m2] + o[m6] + o[m26]);
				k4 = 0.25 * (o[m4] + o[m24] + o[m46] + o[m246]);
				k5 = 0.25 * (o[m0] + o[m2] + o[m4] + o[m24]);
				k6 = 0.25 * (o[m6] + o[m26] + o[m46] + o[m246]);
				k0 = 0.5 * (k1 + k2);


				c1[m] = c1[m] + d1 * k1;
				c2[m] = c2[m] + d1 * k2;
				c3[m] = c3[m] + d2 * k3;
				c4[m] = c4[m] + d2 * k4;


				c0[m] = c0[m] + d1 * k1 + d1 * k2 + d2 * k3 + d2 * k4;


				f[m] = f[m] - (po / dt) * (k1 * (u[m1] + u[m0]) / (2 * dx) - k2 * (u[m2] + u[m0]) / (2 * dx)
					+ k3 * (v[m3] + v[m0]) / (2 * dy) - k4 * (v[m4] + v[m0]) / (2 * dy)
					+ k5 * (w[m5] + w[m0]) / (2 * dz) - k6 * (w[m6] + w[m0]) / (2 * dz));


				if (k5 > k6)
				{
					c0[m] = c0[m] + (k5 - k6) / (dt * dt * dz * g);
					f[m] = f[m] + (k5 - k6) * a[m0] / (dt * dt * dz * g) - (po / dt) * isp * (k5 - k6);
				}

				if (s[m0] > 1)
					f[m] = f[m] - (po / dt) * ((k2 - k1) * u[m0] / dx + (k4 - k3) * v[m0] / dy);

			}


	printf("func2d OK\n");

	omega = 0;//1/(2*d0);

	ptmgs2d(a1, c1, c2, c3, c4, c0, f, s, omega, ep);


	printf("ptm2d OK\n");




	delete[] c0;
	delete[] c1;
	delete[] c2;
	delete[] c3;
	delete[] c4;
	delete[] c5;
	delete[] c6;


	delete[] f;
	delete[] a2;

}

void ptmgs2d(double* u, double* c1, double* c2, double* c3, double* c4,
	double* c0, double* f, int* s, double omega, double e)
{
	double max;
	double tay;
	int i, j, it = 0;
	int m0, m1, m2, m3, m4;


	tay = 2 * omega;
	double RwRw, ww, Aww, Awr, Rr;


	double* r = new double[n2d];
	printf("r2d OK\n");

	//------------------------------------------------------------------------------------------------
	//								����� ����������� �������������(������)
	//------------------------------------------------------------------------------------------------

	do
	{
		//------------------------------------------------------------------------------------------------
		//								�������
		//------------------------------------------------------------------------------------------------

		Aww = 0;
		Awr = 0;
		RwRw = 0;
		ww = 0;

		for (i = 0; i < n1; i++)
			for (j = 0; j < n2; j++)
			{
				m0 = j + n2 * i;

				r[m0] = 0;

			}

		max = 0;

		for (i = 1; i < n1 - 1; i++)
			for (j = 1; j < n2 - 1; j++)
			{
				m0 = j + n2 * i;
				m1 = m0 + n2;
				m2 = m0 - n2;
				m3 = m0 + 1;
				m4 = m0 - 1;



				r[m0] = f[m0] - c0[m0] * u[m0] + (c1[m0] * u[m1] + c2[m0] * u[m2]
					+ c3[m0] * u[m3] + c4[m0] * u[m4]);
				if (max < fabs(r[m0])) { max = fabs(r[m0]); };
			}

		//------------------------------------------------------------------------------------------------
		//								�������� ���������� ����������� �������
		//------------------------------------------------------------------------------------------------


		for (i = 1; i < n1 - 1; i++)
			for (j = 1; j < n2 - 1; j++)
			{
				m0 = j + n2 * i;
				m1 = m0 + n2;
				m2 = m0 - n2;
				m3 = m0 + 1;
				m4 = m0 - 1;

				if (c0[m0] > 0)
				{
					r[m0] = (omega * (c2[m0] * r[m2] + c4[m0] * r[m4]) + r[m0]) / ((0.5 * omega + 1) * c0[m0]);
				}
			}

		for (i = n1 - 2; i > 0; i--)
			for (j = n2 - 2; j > 0; j--)
			{
				m0 = j + n2 * i;
				m1 = m0 + n2;
				m2 = m0 - n2;
				m3 = m0 + 1;
				m4 = m0 - 1;

				if (c0[m0] > 0)
				{
					r[m0] = (omega * (c1[m0] * r[m1] + c3[m0] * r[m3]) + r[m0] * c0[m0]) / ((0.5 * omega + 1) * c0[m0]);
				}
			}

		for (i = 1; i < n1 - 1; i++)

			for (j = 1; j < n2 - 1; j++)
			{
				m0 = j + n2 * i;
				m1 = m0 + n2;
				m2 = m0 - n2;
				m3 = m0 + 1;
				m4 = m0 - 1;

				if (c0[m0] > 0)
				{

					Awr = c0[m0] * r[m0] - (c1[m0] * r[m1] + c2[m0] * r[m2] + c3[m0] * r[m3] + c4[m0] * r[m4]);
					Rr = 0.5 * c0[m0] * r[m0] - (c1[m0] * r[m1] + c3[m0] * r[m3]);

					RwRw = RwRw + Rr * Rr / c0[m0];
					Aww = Aww + Awr * r[m0];
					ww = ww + c0[m0] * r[m0] * r[m0];
				}
			}

		if (ww > 0)
		{
			tay = 2 * omega + ww / Aww;
			omega = sqrt(ww / RwRw);
		}
		omega = 0;
		//------------------------------------------------------------------------------------------------
		//								��� ���
		//------------------------------------------------------------------------------------------------

		for (m0 = 0; m0 < n2d; m0++)
			u[m0] = u[m0] + tay * r[m0];

		it++;


		printf("%f	%f	%f	%f	%d\n", t, max * 1000000, omega, tay, it);
	} while ((max > e) && (it < 10000));


	delete[] r;


	//------------------------------------------------------------------------------------------------
	//								����� ����������� �������������(�����)
	//------------------------------------------------------------------------------------------------

}

//------------------------------------------------------------------------------------------------
//								�����
//------------------------------------------------------------------------------------------------



void ptmsp(double* u, double* c1, double* c2, double* c3, double* c4,
	double* c5, double* c6, double* c01, double* c02, double* c0,
	double* f, int* s, double omega, double e)
{
	double max;
	double tay, alf = 1, b = 0;
	int i, j, k, it = 0;
	int m0, m1, m2, m3, m4, m5, m6;



	double AwAw, ww, Aww, Aw1w1, B1AwAw, B1AwAw1 = 1;


	double* r = new double[n];
	double* wr = new double[n];
	double* Awr = new double[n];
	double* Aksi = new double[n];
	double* ksi = new double[n];

	//------------------------------------------------------------------------------------------------
	//								����� ����������� �������������(������)
	//------------------------------------------------------------------------------------------------
	for (m0 = 0; m0 < n; m0++)
	{
		ksi[m0] = 0;
	}

	do
	{
		//------------------------------------------------------------------------------------------------
		//								�������
		//------------------------------------------------------------------------------------------------

		AwAw = 0;
		ww = 0;

		Aww = 0;

		B1AwAw = 0;

		for (m0 = 0; m0 < n; m0++)
		{
			r[m0] = 0;
			wr[m0] = 0;
			Awr[m0] = 0;
		}

		max = 0;

		for (k = 1; k < n3; k++)
			for (i = 1; i < n1; i++)
				for (j = 1; j < n2; j++)
				{
					m0 = j + (n2 + 1) * (i + (n1 + 1) * k);
					m1 = m0 + (n2 + 1);
					m2 = m0 - (n2 + 1);
					m3 = m0 + 1;
					m4 = m0 - 1;
					m5 = m0 + (n2 + 1) * (n1 + 1);
					m6 = m0 - (n2 + 1) * (n1 + 1);

					if (s[m0] != 0)
					{
						r[m0] = f[m0] - c0[m0] * u[m0] + (c1[m0] * u[m1] + c2[m0] * u[m2]
							+ c3[m0] * u[m3] + c4[m0] * u[m4] + c5[m0] * u[m5] + c6[m0] * u[m6]);
						if (max < fabs(r[m0])) { max = fabs(r[m0]); };
					}
				}

		//------------------------------------------------------------------------------------------------
		//								�������� ���������� ����������� �������
		//------------------------------------------------------------------------------------------------


		for (k = 1; k < n3; k++)
			for (i = 1; i < n1; i++)
				for (j = 1; j < n2; j++)
				{
					m0 = j + (n2 + 1) * (i + (n1 + 1) * k);
					m2 = m0 - (n2 + 1);
					m4 = m0 - 1;
					m6 = m0 - (n2 + 1) * (n1 + 1);

					if (s[m0] != 0)
					{
						wr[m0] = (omega * (c2[m0] * wr[m2] + c4[m0] * wr[m4] + c6[m0] * wr[m6]) + r[m0]) / (omega * c0[m0] / 2 + 1);
					}
				}

		for (k = n3 - 1; k > 0; k--)
			for (i = n1 - 1; i > 0; i--)
				for (j = n2 - 1; j > 0; j--)
				{
					m0 = j + (n2 + 1) * (i + (n1 + 1) * k);
					m1 = m0 + (n2 + 1);
					m3 = m0 + 1;
					m5 = m0 + (n2 + 1) * (n1 + 1);

					if (s[m0] != 0)
					{
						wr[m0] = (omega * (c1[m0] * wr[m1] + c3[m0] * wr[m3] + c5[m0] * wr[m5]) + wr[m0]) / (omega * c0[m0] / 2 + 1);
					}
				}

		for (k = 1; k < n3; k++)
			for (i = 1; i < n1; i++)
				for (j = 1; j < n2; j++)
				{
					m0 = j + (n2 + 1) * (i + (n1 + 1) * k);
					m1 = m0 + (n2 + 1);
					m2 = m0 - (n2 + 1);
					m3 = m0 + 1;
					m4 = m0 - 1;
					m5 = m0 + (n2 + 1) * (n1 + 1);
					m6 = m0 - (n2 + 1) * (n1 + 1);

					if (s[m0] != 0)
					{

						Awr[m0] = c0[m0] * wr[m0] - (c1[m0] * wr[m1] + c2[m0] * wr[m2] + c3[m0] * wr[m3] + c4[m0] * wr[m4] + c5[m0] * wr[m5] + c6[m0] * wr[m6]);

						ww = ww + wr[m0] * wr[m0];
						AwAw = Aww + Awr[m0] * Awr[m0];

						Aww = Aww + Awr[m0] * wr[m0];
					}
				}

		if (it > 0)
			b = Aww / Aw1w1;

		Aw1w1 = Aww;

		for (m0 = 0; m0 < n; m0++)
		{
			ksi[m0] = wr[m0] + b * ksi[m0];
		}



		for (k = 1; k < n3; k++)
			for (i = 1; i < n1; i++)
				for (j = 1; j < n2; j++)
				{
					m0 = j + (n2 + 1) * (i + (n1 + 1) * k);
					m1 = m0 + (n2 + 1);
					m2 = m0 - (n2 + 1);
					m3 = m0 + 1;
					m4 = m0 - 1;
					m5 = m0 + (n2 + 1) * (n1 + 1);
					m6 = m0 - (n2 + 1) * (n1 + 1);

					if (s[m0] != 0)
					{

						Aksi[m0] = c0[m0] * ksi[m0] - (c1[m0] * ksi[m1] + c2[m0] * ksi[m2] + c3[m0] * ksi[m3] + c4[m0] * ksi[m4] + c5[m0] * ksi[m5] + c6[m0] * ksi[m6]);

					}
				}


		for (k = 1; k < n3; k++)
			for (i = 1; i < n1; i++)
				for (j = 1; j < n2; j++)
				{
					m0 = j + (n2 + 1) * (i + (n1 + 1) * k);
					m2 = m0 - (n2 + 1);
					m4 = m0 - 1;
					m6 = m0 - (n2 + 1) * (n1 + 1);

					if (s[m0] != 0)
					{
						r[m0] = (omega * (c2[m0] * r[m2] + c4[m0] * r[m4] + c6[m0] * r[m6]) + Aksi[m0]) / (omega * c0[m0] / 2 + 1);
					}
				}

		for (k = n3 - 1; k > 0; k--)
			for (i = n1 - 1; i > 0; i--)
				for (j = n2 - 1; j > 0; j--)
				{
					m0 = j + (n2 + 1) * (i + (n1 + 1) * k);
					m1 = m0 + (n2 + 1);
					m3 = m0 + 1;
					m5 = m0 + (n2 + 1) * (n1 + 1);

					if (s[m0] != 0)
					{
						r[m0] = (omega * (c1[m0] * r[m1] + c3[m0] * r[m3] + c5[m0] * r[m5]) + r[m0]) / (omega * c0[m0] / 2 + 1);
						B1AwAw = B1AwAw + r[m0] * Aksi[m0];
					}
				}


		omega = 2 * sqrt(ww / AwAw);
		tay = Aww / B1AwAw;



		//------------------------------------------------------------------------------------------------
		//								��� ���
		//------------------------------------------------------------------------------------------------


		for (m0 = 0; m0 < n; m0++)
		{
			u[m0] = u[m0] + tay * ksi[m0];
		}

		it++;

		printf("%f	%f	%f	%f	%d\n", t, max * 1000000, omega, tay, it);
	} while ((max > e) && (it < 1000));

	delete[] r;
	delete[] wr;
	delete[] Awr;
	delete[] ksi;
	delete[] Aksi;

	//------------------------------------------------------------------------------------------------
	//								����� ����������� �������������(�����)
	//------------------------------------------------------------------------------------------------

}



void ptmgs(double* u, double* c1, double* c2, double* c3, double* c4,
	double* c5, double* c6, double* c01, double* c02, double* c0,
	double* f, int* s, double omega, double e)
{
	double max;
	double tay;
	int i, j, k, it = 0;
	int m0, m1, m2, m3, m4, m5, m6;



	double BAwAw, BAw1Aw1, wBAw, Rwr, RBAwr, RwRBAw, Aww, ss, kk, rw, tetta;

	FILE* rez;

	rez = fopen("ss1.txt", "w+");

	double* r = new double[n];
	double* wr = new double[n];
	double* wr1 = new double[n];
	double* Awr = new double[n];
	double* Awr1 = new double[n];
	double* BAwr = new double[n];
	double* BAwr1 = new double[n];

	//------------------------------------------------------------------------------------------------
	//								����� ����������� �������������(������)
	//------------------------------------------------------------------------------------------------

	do
	{
		//------------------------------------------------------------------------------------------------
		//								�������
		//------------------------------------------------------------------------------------------------

		Aww = 0;
		BAwAw = 0;
		BAw1Aw1 = 0;
		ss = 1;
		kk = 0;
		rw = 0;
		tetta = 1;
		wBAw = 0;
		Rwr = 0;
		RBAwr = 0;
		RwRBAw = 0;

		for (m0 = 0; m0 < n; m0++)
		{
			r[m0] = 0;
			wr[m0] = 0;
			wr1[m0] = 0;
			Awr[m0] = 0;

			BAwr[m0] = 0;
			Awr1[m0] = 0;
			BAwr1[m0] = 0;
		}

		max = 0;

		for (k = 1; k < n3; k++)
			for (i = 1; i < n1; i++)
				for (j = 1; j < n2; j++)
				{
					m0 = j + (n2 + 1) * (i + (n1 + 1) * k);
					m1 = m0 + (n2 + 1);
					m2 = m0 - (n2 + 1);
					m3 = m0 + 1;
					m4 = m0 - 1;
					m5 = m0 + (n2 + 1) * (n1 + 1);
					m6 = m0 - (n2 + 1) * (n1 + 1);

					if (s[m0] != 0)
					{
						r[m0] = f[m0] - c0[m0] * u[m0] + (c1[m0] * u[m1] + c2[m0] * u[m2]
							+ c3[m0] * u[m3] + c4[m0] * u[m4] + c5[m0] * u[m5] + c6[m0] * u[m6]);
						if (max < fabs(r[m0])) { max = fabs(r[m0]); };
					}
				}

		//------------------------------------------------------------------------------------------------
		//								�������� ���������� ����������� �������
		//------------------------------------------------------------------------------------------------


		for (k = 1; k < n3; k++)
			for (i = 1; i < n1; i++)
				for (j = 1; j < n2; j++)
				{
					m0 = j + (n2 + 1) * (i + (n1 + 1) * k);
					m1 = m0 + (n2 + 1);
					m2 = m0 - (n2 + 1);
					m3 = m0 + 1;
					m4 = m0 - 1;
					m5 = m0 + (n2 + 1) * (n1 + 1);
					m6 = m0 - (n2 + 1) * (n1 + 1);

					if (s[m0] != 0)
					{
						wr[m0] = (omega * (0.5 * (c2[m0] + c1[m2]) * wr[m2] + 0.5 * (c4[m0] + c3[m4]) * wr[m4] + 0.5 * (c6[m0] + c5[m6]) * wr[m6]) + r[m0]) / (0.5 * omega * c0[m0] + 1);
					}
				}

		for (k = n3 - 1; k > 0; k--)
			for (i = n1 - 1; i > 0; i--)
				for (j = n2 - 1; j > 0; j--)
				{
					m0 = j + (n2 + 1) * (i + (n1 + 1) * k);
					m1 = m0 + (n2 + 1);
					m2 = m0 - (n2 + 1);
					m3 = m0 + 1;
					m4 = m0 - 1;
					m5 = m0 + (n2 + 1) * (n1 + 1);
					m6 = m0 - (n2 + 1) * (n1 + 1);

					if (s[m0] != 0)
					{
						wr[m0] = (omega * (0.5 * (c1[m0] + c2[m1]) * wr[m1] + 0.5 * (c3[m0] + c4[m3]) * wr[m3] + 0.5 * (c5[m0] + c6[m5]) * wr[m5]) + wr[m0]) / (0.5 * omega * c0[m0] + 1);
					}
				}

		for (k = 1; k < n3; k++)
			for (i = 1; i < n1; i++)
				for (j = 1; j < n2; j++)
				{
					m0 = j + (n2 + 1) * (i + (n1 + 1) * k);
					m1 = m0 + (n2 + 1);
					m2 = m0 - (n2 + 1);
					m3 = m0 + 1;
					m4 = m0 - 1;
					m5 = m0 + (n2 + 1) * (n1 + 1);
					m6 = m0 - (n2 + 1) * (n1 + 1);

					if (s[m0] != 0)
					{

						Awr[m0] = c0[m0] * wr[m0] - (0.5 * (c1[m0] + c2[m1]) * wr[m1] + 0.5 * (c2[m0] + c1[m2]) * wr[m2] + 0.5 * (c3[m0] + c4[m3]) * wr[m3] +
							0.5 * (c4[m0] + c3[m4]) * wr[m4] + 0.5 * (c5[m0] + c6[m5]) * wr[m5] + 0.5 * (c6[m0] + c5[m6]) * wr[m6]);
						wr1[m0] = c0[m0] * r[m0] - (0.5 * (c1[m0] + c2[m1]) * r[m1] + 0.5 * (c2[m0] + c1[m2]) * r[m2] + 0.5 * (c3[m0] + c4[m3]) * r[m3] +
							0.5 * (c4[m0] + c3[m4]) * r[m4] + 0.5 * (c5[m0] + c6[m5]) * r[m5] + 0.5 * (c6[m0] + c5[m6]) * r[m6]);
						Awr1[m0] = -(0.5 * (c1[m0] - c2[m1]) * wr[m1] + 0.5 * (c2[m0] - c1[m2]) * wr[m2] + 0.5 * (c3[m0] - c4[m3]) * wr[m3] +
							0.5 * (c4[m0] - c3[m4]) * wr[m4] + 0.5 * (c5[m0] - c6[m5]) * wr[m5] + 0.5 * (c6[m0] - c5[m6]) * wr[m6]);
					}
				}

		for (k = 1; k < n3; k++)
			for (i = 1; i < n1; i++)
				for (j = 1; j < n2; j++)
				{
					m0 = j + (n2 + 1) * (i + (n1 + 1) * k);
					m1 = m0 + (n2 + 1);
					m2 = m0 - (n2 + 1);
					m3 = m0 + 1;
					m4 = m0 - 1;
					m5 = m0 + (n2 + 1) * (n1 + 1);
					m6 = m0 - (n2 + 1) * (n1 + 1);

					if (s[m0] != 0)
					{
						BAwr[m0] = (omega * (0.5 * (c2[m0] + c1[m2]) * BAwr[m2] + 0.5 * (c4[m0] + c3[m4]) * BAwr[m4] + 0.5 * (c6[m0] + c5[m6]) * BAwr[m6]) + Awr[m0]) / (0.5 * omega * c0[m0] + 1);
						BAwr1[m0] = (omega * (0.5 * (c2[m0] + c1[m2]) * BAwr1[m2] + 0.5 * (c4[m0] + c3[m4]) * BAwr1[m4] + 0.5 * (c6[m0] + c5[m6]) * BAwr1[m6]) + Awr1[m0]) / (0.5 * omega * c0[m0] + 1);
					}
				}

		for (k = n3 - 1; k > 0; k--)
			for (i = n1 - 1; i > 0; i--)
				for (j = n2 - 1; j > 0; j--)
				{
					m0 = j + (n2 + 1) * (i + (n1 + 1) * k);
					m1 = m0 + (n2 + 1);
					m2 = m0 - (n2 + 1);
					m3 = m0 + 1;
					m4 = m0 - 1;
					m5 = m0 + (n2 + 1) * (n1 + 1);
					m6 = m0 - (n2 + 1) * (n1 + 1);

					if (s[m0] != 0)
					{
						BAwr[m0] = (omega * (0.5 * (c1[m0] + c2[m1]) * BAwr[m1] + 0.5 * (c3[m0] + c4[m3]) * BAwr[m3] + 0.5 * (c5[m0] + c6[m5]) * BAwr[m5]) + BAwr[m0]) / (0.5 * omega * c0[m0] + 1);
						BAwr1[m0] = (omega * (0.5 * (c1[m0] + c2[m1]) * BAwr1[m1] + 0.5 * (c3[m0] + c4[m3]) * BAwr1[m3] + 0.5 * (c5[m0] + c6[m5]) * BAwr1[m5]) + BAwr1[m0]) / (0.5 * omega * c0[m0] + 1);

					}
				}

		for (k = 1; k < n3; k++)
			for (i = 1; i < n1; i++)
				for (j = 1; j < n2; j++)
				{
					m0 = j + (n2 + 1) * (i + (n1 + 1) * k);
					m1 = m0 + (n2 + 1);
					m2 = m0 - (n2 + 1);
					m3 = m0 + 1;
					m4 = m0 - 1;
					m5 = m0 + (n2 + 1) * (n1 + 1);
					m6 = m0 - (n2 + 1) * (n1 + 1);

					if (s[m0] != 0)
					{
						wBAw = wBAw + wr[m0] * BAwr[m0];
						Rwr = 0.5 * c0[m0] * wr[m0] - (0.5 * (c2[m0] + c1[m2]) * wr[m2] +
							0.5 * (c4[m0] + c3[m4]) * wr[m4] + 0.5 * (c6[m0] + c5[m6]) * wr[m6]);
						RBAwr = 0.5 * c0[m0] * BAwr[m0] - (0.5 * (c2[m0] + c1[m2]) * BAwr[m2] +
							0.5 * (c4[m0] + c3[m4]) * BAwr[m4] + 0.5 * (c6[m0] + c5[m6]) * BAwr[m6]);
						RwRBAw = RwRBAw + Rwr * RBAwr;

						rw = rw + r[m0] * wr[m0];
						Aww = Aww + Awr[m0] * wr[m0];
						BAwAw = BAwAw + BAwr[m0] * Awr[m0];
						BAw1Aw1 = BAw1Aw1 + BAwr1[m0] * Awr1[m0];
					}
				}



		omega = 0;
		if ((wBAw / RwRBAw) > 0)
			omega = sqrt(fabs(wBAw / RwRBAw));
		tay = Aww / BAwAw;
		ss = sqrt(1 - Aww * Aww / (BAwAw * rw));
		kk = (BAw1Aw1 / rw) * (Aww / BAwAw) * (Aww / BAwAw);
		tetta = (1 - ss * sqrt(kk / (1 + kk - ss * ss))) / (1 + kk);
		tay = tay * tetta;



		//------------------------------------------------------------------------------------------------
		//								��� ���
		//------------------------------------------------------------------------------------------------


		for (m0 = 0; m0 < n; m0++)
			u[m0] = u[m0] + tay * wr[m0];

		it++;

		printf("%f	%f	%f	%f	%d\n", t, max * 1000000, omega, tetta, it);
		fprintf(rez, "%f	%f	%f	%f	%f	%f	%f	%d\n", t, max * 1000000, kk, ss, tetta, omega, tay, it);
	} while ((max > e) && (it < 200));

	fclose(rez);

	getch();

	delete[] r;
	delete[] wr;
	delete[] wr1;
	delete[] Awr;
	delete[] BAwr;
	delete[] Awr1;
	delete[] BAwr1;
	//------------------------------------------------------------------------------------------------
	//								����� ����������� �������������(�����)
	//------------------------------------------------------------------------------------------------

}
