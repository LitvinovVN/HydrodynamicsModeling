#include "hip/hip_runtime.h"
#ifndef CUDA_DEVICES_INFO_CU
#define CUDA_DEVICES_INFO_CU

#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <sstream> // std::ostringstream
#include "CUDA_Devices_Info.h"

extern "C"
void printGpuParameters(std::string prefixDescr) {	
	int deviceCount;
	hipDeviceProp_t devProp;

	hipGetDeviceCount(&deviceCount);
	
	std::ostringstream ss;
	ss << "------- Printing CUDA-compatible device properties -------" << std::endl;
    ss << prefixDescr << std::endl;
	ss << "Found " << deviceCount << " cuda compatible devices" << std::endl;

	for(int device = 0; device < deviceCount; device++){
		hipGetDeviceProperties(&devProp, device);

		ss << "  --- Device " << device << " ---" << std::endl;
		ss << "Compute capability              : " << devProp.major << "." << devProp.minor << std::endl;
		ss << "Name                            : " << devProp.name << std::endl;
		ss << "Total Global Memory             : " << devProp.totalGlobalMem << " bytes ("<< devProp.totalGlobalMem  / 1024.0 / 1024.0 / 1024.0 << " Gb)" << std::endl;
		ss << "Shared memory per block         : " << devProp.sharedMemPerBlock << " bytes" << std::endl;
		ss << "Shared memory per multiprocessor: " << devProp.sharedMemPerMultiprocessor << " bytes" << std::endl;
		ss << "Registers per block             : " << devProp.regsPerBlock << std::endl;
		ss << "Registers per multiprocessor    : " << devProp.regsPerMultiprocessor << std::endl;
		ss << "Warp size                       : " << devProp.warpSize << std::endl;
		ss << "Max threads per block           : " << devProp.maxThreadsPerBlock << std::endl;
		ss << "Total constant memory           : " << devProp.totalConstMem << " bytes" << std::endl;
		ss << "Clock rate                      : " << devProp.clockRate << " kHz" << std::endl;
		ss << "Global memory bus width         : " << devProp.memoryBusWidth << " bits" << std::endl;
		ss << "Peak memory clock frequency     : " << devProp.memoryClockRate << " kHz" << std::endl;
		ss << "Texture alignment               : " << devProp.textureAlignment << std::endl;
		ss << "Device overlap                  : " << devProp.deviceOverlap << std::endl;
		ss << "Multiprocessor count            : " << devProp.multiProcessorCount << std::endl;
		ss << "Max threads dim                 : " << devProp.maxThreadsDim[0] << " " << devProp.maxThreadsDim[1] << " " << devProp.maxThreadsDim[2] << std::endl;
		ss << "Max threads per block           : " << devProp.maxThreadsPerBlock << std::endl;
		ss << "Max threads per multiprocessor  : " << devProp.maxThreadsPerMultiProcessor << std::endl;
		ss << "Max grid num                    : " << devProp.maxGridSize[0] << " " << devProp.maxGridSize[1] << " " << devProp.maxGridSize[2] << std::endl;
	}

    std::cout << ss.str();
}

#endif